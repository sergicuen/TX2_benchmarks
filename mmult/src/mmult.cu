/////////////////////////////////////////////////////////////////////////////  
// 
//	file: mmult.cu
//  versi�n para radiar en CNA
//	usage: mat_mul_redundant -s SIZE -x threads_per_block_x -y threads_per_block_y -b replicate_input -r RunsBlock
//
/////////////////////////////////////////////////////////////////////////////  
//	Descripci�n: Genera 2 kernels concurrentes que realizan C=AxB
//  - if (duplicate_input==1)
//			kernel_1: C = A x B
//			kernel_2: C'= A' x B'
//			Check_DMR: check(C==C')
//  - else
//			kernel_1: C = A x B
//			kernel_2: C'= A x B
//			Check_DMR: check(C==C')
//
//  M�dulos a�adidos:
//  - VarInit: inicializa las matrices con valores conocidos para que el resultado 
//		de AxB pueda ser verificado por la CPU de forma r�pida
//  - check_golden: genera los resultados C_golden con la CPU y comprueba si 
//		coinciden con los resultados GPU
//  - c�lculo del CRC de los resultados (todo)
//  
//	Funcionalidad adicional:
//  - A�adida verificaci�n adicional:
//		if (check_DMR=correct) then 
//			if (check_golden ==correct) then Error_Masked
//			else SDC_undetected
//		else SDC_detected
//
////////////////////////////////////////////////////////////////////////////////////////////
//
//  TODO: 
//	- verificar que lo c�lculos se realizan aunque los datos de entrada no hayan cambiado
//  - bajar los clocks
//  - SOLO FUNCIONA CON NUMBER_OF_STREAMS 2


#include "stdio.h"
#include <hip/hip_runtime.h>

#include <unistd.h>
#include<sys/time.h>

#define NUMBER_OF_STREAMS 2
#define USE_GOLDEN 0    // inicializa mat para comprobaci�n con golden en CPU
#define PERFORMANCE 0   // muestra tiempos de ejecuci�n
#define ROBUST_PRINTING 0 // muestra detalles de errores
#define INJECT_FAULTS 0
//#define RUNBLOCK 1000  // par�metro de entrada

using namespace std;

unsigned int runs_counter=0;
unsigned int runs_werror=0;

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "-run: %d; %s in %s at line %d\n", runs_counter, hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void matrixMultiplicationKernel(void *A, void *B, void *C, int N){
    
    int ROW = (int) (blockIdx.y * blockDim.y + threadIdx.y);
    int COL = (int) (blockIdx.x * blockDim.x + threadIdx.x);
    
    float * ptr_A = ((float *)A);
    float * ptr_B = ((float *)B);
    float * ptr_C = ((float *)C);

    float tmpSum = 0;

    if (ROW < N && COL < N) {
//#pragma unroll 1
        for (int i = 0; i < N; ++i) {
            tmpSum += ptr_A[ROW * N + i] * ptr_B[i * N + COL];
        }
        ptr_C[ROW * N + COL] = tmpSum;
    }
}

void initMatrices( void *h_A, void *h_B, int N ){
  // Initialize matrices on the host
  for (int i = 0; i < N; ++i){
    for (int j = 0; j < N; ++j){
    ((float*)h_A)[i*N+j] = i;
    ((float*)h_B)[i*N+j] = N-i;
    }
  }
}

//Inicializaci�n de matrices para check_error en CPU
void initMat4Golden(void *data, int n, int m) {
    int ct = 0;
    float val, tmp;
    for (int i = 1; i < n+1; ++i) {
      val = i;
      for (int j = 1; j < m+1; ++j) {
         tmp= val/(n*m);
		 ((float*)data)[ct++]=tmp;
        val += i;
      }
    }
}

void inject_faults(void *h_C, int N){
  ((float*)h_C)[2*N+1] = 7.5;
  ((float*)h_C)[N] = 7.5;
}


void check_and_parse(int argc, char* argv[] , int * N , int * threads_per_blockx , int * threads_per_blocky, int * rblock, bool * replicate_input){

  if (argc !=9  && argc !=11){
    printf("usage: mat_mul_redundant -s SIZE -x threads_per_block_x -y threads_per_block_y -b replicate_input -r RunsBlock\n");
    exit(EXIT_FAILURE);
  }

  int opt;

  while ((opt = getopt(argc, argv, "s:x:y:r:b")) != -1){
    switch (opt) {
      case 's':
        if(optarg) *N = (unsigned long long) atoi(optarg);
        break;
      case 'x': 
        if(optarg) *threads_per_blockx = atoi(optarg);
        break;
      case 'y': 
        if(optarg) *threads_per_blocky = atoi(optarg);
        break;
      case 'r':
        if(optarg) *rblock = (unsigned int) atoi(optarg);
        break;
      case 'b':
        if(optarg) *replicate_input = (atoi(optarg) == 1);
        break;

      default: /* '?' */
        
        exit(EXIT_FAILURE);
    }
  }
}


void mem_alloc_host(void ** h_A, void ** h_B , void ** h_C1, void ** h_C2,  int  SIZE){
  
  HANDLE_ERROR( hipHostAlloc( h_A, SIZE, hipHostMallocDefault ) );
  HANDLE_ERROR( hipHostAlloc( h_B, SIZE, hipHostMallocDefault ) );
  HANDLE_ERROR( hipHostAlloc( h_C1, SIZE, hipHostMallocDefault ) );
  HANDLE_ERROR( hipHostAlloc( h_C2, SIZE, hipHostMallocDefault ) );
}

void mem_alloc_device(void ** d_A1, void ** d_B1,void ** d_C1, void ** d_C2,void ** d_A2, void ** d_B2,  bool replicate_input , int  SIZE , void ** d_pA, void ** d_pB){
  
  HANDLE_ERROR( hipMalloc( d_A1, SIZE ) );
  HANDLE_ERROR( hipMalloc( d_B1, SIZE ) );
  HANDLE_ERROR( hipMalloc( d_C1, SIZE ) );
  HANDLE_ERROR( hipMalloc( d_C2, SIZE ) );
  if (replicate_input){
    HANDLE_ERROR( hipMalloc( d_A2, SIZE ) );
    HANDLE_ERROR( hipMalloc( d_B2, SIZE ) );
  }
  d_pA[0] = d_A1[0];
  d_pB[0] = d_B1[0] ; 
  if (replicate_input){
    d_pA[0] = d_A2[0] ; 
    d_pB[0] = d_B2[0] ;
  }

}

void mem_free_host(void *h_A, void *h_B, void *h_C1, void *h_C2){
  HANDLE_ERROR( hipHostFree( h_A));
  HANDLE_ERROR( hipHostFree( h_B));
  HANDLE_ERROR( hipHostFree( h_C1));
  HANDLE_ERROR( hipHostFree( h_C2));
}

void mem_free_device(void * d_A1,void * d_B1,void *d_A2,void *d_B2 ,void * d_C1 ,void * d_C2 , bool replicate_input){
 
  HANDLE_ERROR( hipFree( d_A1 ) );
  HANDLE_ERROR( hipFree( d_B1 ) );

  if (replicate_input){
  HANDLE_ERROR( hipFree( d_A2 ) );
  HANDLE_ERROR( hipFree( d_B2 ) );
  }

  HANDLE_ERROR( hipFree( d_C1 ) );
  HANDLE_ERROR( hipFree( d_C2 ) );
}


void mem_copy_to_device(void * d_A1 , void * d_A2, void * d_B1 , void * d_B2 ,  void * h_A , void * h_B, size_t SIZE, bool replicate_input , hipStream_t *  stream  ){
  
  HANDLE_ERROR(hipMemcpyAsync(d_A1,h_A,SIZE,hipMemcpyHostToDevice,stream[0]));
  HANDLE_ERROR(hipMemcpyAsync(d_B1,h_B,SIZE,hipMemcpyHostToDevice,stream[0]));

  if (replicate_input){
    HANDLE_ERROR( hipMemcpyAsync( d_A2, h_A,SIZE, hipMemcpyHostToDevice, stream[1]) );
    HANDLE_ERROR( hipMemcpyAsync( d_B2, h_B,SIZE, hipMemcpyHostToDevice, stream[1]) );
    }
}


void mem_copy_to_host(void * h_C1,void * h_C2,  void *  d_C1, void * d_C2,  size_t SIZE ,hipStream_t *  stream ){
    
  HANDLE_ERROR( hipMemcpyAsync( h_C1, d_C1,SIZE, hipMemcpyDeviceToHost, stream[0] ));
 if(NUMBER_OF_STREAMS==2){ HANDLE_ERROR( hipMemcpyAsync( h_C2, d_C2,SIZE, hipMemcpyDeviceToHost, stream[1] ));}

}

//Revisa toda la matriz y env�a n�mero de errores detectados
bool test( void * h_C1, void * h_C2, int N){
  bool correct = true ;
  int   local_errors=0;
  for (int i = 0; i < N ; ++i){
    for (int j = 0; j < N; ++j){
      correct = (((float*)h_C1)[j + i*N] == ((float*)h_C2)[j + i*N]);
	  if (correct == false){
          local_errors++;
          #if ROBUST_PRINTING
            printf("ERROR detected at C[%d][%d]\n", i, j);
          #endif
      }
    }
  }
  if (local_errors !=0) {
      printf("- #run: %u\n", runs_counter);
	  printf("ERRORS detected: %d\n", local_errors);
      correct=false;
  }

  return correct;
}

// Revisa  la matriz y salta al primer error
bool FindFirstError( void * h_C1, void * h_C2, int N){
  bool correct = true ;
  
  for (int i = 0; i < N && correct==true; ++i){
    for (int j = 0; j < N && correct==true; ++j){
      correct = (((float*)h_C1)[j + i*N] == ((float*)h_C2)[j + i*N]);
	  if (correct == false){
          printf("- #run: %u\n", runs_counter);
          printf("ERROR detected at C[%d][%d]\n", i, j);
      }
    }
  }

  return correct;
}



// check_error en CPU
bool Golden_check(void *h_C, int m, int n, int k) {
    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6 ; // machine zero
//    double eps = 1.e-10 ; // machine zero
    bool correct = true;
    unsigned int local_errors=0;
//    const float factor = ( 1.0f*n * 1.0f*(n+1) * 1.0f*(2*n+1) )/ (6.0f*k*k);
    const float factor = ( 1.0f*n * 1.0f*(n+1) * 1.0f*(2*n+1) )/ (6.0f*k*k*k*k);
    int ct = 0;
    for (int i = 1; i < m+1; i++) {
      for (int j = 1; j < n+1; j++) {
        double abs_err = fabs(((float*)h_C)[ct] - (i*j*factor));
        double dot_length = k;
        double abs_val = fabs(((float*)h_C)[ct]);
        double rel_err = abs_err/abs_val/dot_length;
		
        if (rel_err > eps)
        {
            // printf("- #run: %u\n", runs_counter);            
            // printf("ERROR: Matrix[%d][%d]=%.8f, ref=%.8f error term is > %E\n", 
               // i-1, j-1, ((float*)h_C)[ct], i*j*factor, eps);
            // correct = false;
          local_errors++;

        }
        ct++;
      }
    }
    if (local_errors !=0){
      printf("- #run: %u\n", runs_counter); 
      printf(" ERRORS: %u\n", local_errors);  
      correct=false;
    }
    return correct;
}


void printMatrix( void * matrix , int N){
  for(int i = 0 ; i < N ; i++){
    for(int j=0; j<N ; j++){
      printf("%i ", (int)((float *)matrix)[i * N + j ]);
	  //printf("%.8f ", ((float *)matrix)[i * N + j ]);
    }
    printf("\n");
  }
}

int main (int argc, char* argv[]) {

  /*
  * Data
  */
  int N;
  int threads_per_blockx;
  int threads_per_blocky;
  struct timeval time_start;
  struct timeval time_end;
  struct timeval time_compare1;
  struct timeval time_compare2;
  unsigned int TotalexTime;

  bool replicate_input = true;
  int rblock=10;
  bool correct_C1=true;
  // bool correct_C2=true;
  bool DMR_correct=true;
  /*
  * Checking arguments used to call the program
  */
  check_and_parse(argc,argv,&N,&threads_per_blockx,&threads_per_blocky,&rblock,&replicate_input);
  #if ROBUST_PRINTING
    printf("Hw: Jetson TX2, Pascal arch \r\n");
    printf("Test: MMULT_DMR\r\n");
    if (replicate_input) printf("Replicating the input for each kernel\n");
    printf("Version: 1.0 \r\n");
    printf("matrix size:%d\n", N);  
    printf(" data type: float\r\n");
  #else
    printf("Hw:TX2, T:MMULT_DMR, V:%d, ThBlck:%d, Sz:%d, Dt:fp, RunsB:%d\r\n", replicate_input, threads_per_blockx, N, rblock);
  #endif
  
  /*
  * Perform matrix multiplication C = A*B
  * where A, B and C are NxN matrices
  * The matrices are floating point values
  */ 
  int SIZE = (int)N*N;
  if(SIZE < 0){ printf("Size Overflow\n"); exit(EXIT_FAILURE);}
  //printf("Matrix of size: %d\n",SIZE);

gettimeofday(&time_start, NULL);

  /*
  * Allocate pinned memory on the CPU to make asynchronous transfers.
  */
  void *h_A, *h_B, *h_C1, *h_C2;
  mem_alloc_host(&h_A,&h_B,&h_C1,&h_C2, SIZE * sizeof(float));

  /*
  * Initialize matrices on the host
  */
#if USE_GOLDEN
  initMat4Golden(h_A, N, N);
  initMat4Golden(h_B, N, N);
#else
  if (NUMBER_OF_STREAMS==1) {initMat4Golden(h_A, N, N); initMat4Golden(h_B, N, N);}
  else initMatrices(h_A,h_B,N);
#endif

runs_counter=0;

while (1){
//for (runs_counter=0; runs_counter < rblock; runs_counter++){


    //printf("MaT_A\n");
    //printMatrix( h_A,  N);

  /*
  * Allocate memory on the device. 
  * Create pointers to data structure
  * Create two temporary pointers for replicate input purpose
  */
  void *d_A1, *d_B1, *d_A2, *d_B2, *d_C1, *d_C2, *d_pA , *d_pB;
  mem_alloc_device(&d_A1,&d_B1,&d_C1,&d_C2,&d_A2,&d_B2,replicate_input, SIZE * sizeof(float),&d_pA,&d_pB);
  
  /*
  * Setting the block, grid dimension for GPU computation
  */
  int blocks_count_x = (N + threads_per_blockx - 1)/ threads_per_blockx;
  int blocks_count_y = (N + threads_per_blocky - 1)/ threads_per_blocky;  
  dim3 threadsPerBlock(threads_per_blockx, threads_per_blocky);
  dim3 blocksPerGrid(blocks_count_x, blocks_count_y);

  /*
  * Creating the streams for the kernel launch
  */
  hipStream_t stream[NUMBER_OF_STREAMS];
  for (int i = 0; i < NUMBER_OF_STREAMS; ++i)
    HANDLE_ERROR( hipStreamCreate(&stream[i]));

  /*
  * Copy the arrays 'a' and 'b' to the GPU
  */
  mem_copy_to_device(d_A1,d_A2,d_B1,d_B2,h_A,h_B,SIZE * sizeof(float),replicate_input,&stream[0]);

  /*
  * Sync streams 
  */
  HANDLE_ERROR (hipStreamSynchronize(stream[0]) );
  if (NUMBER_OF_STREAMS==2){ HANDLE_ERROR (hipStreamSynchronize(stream[1]) );}
 // gettimeofday(&time_start, NULL);

  /*
  * Kernels launch, second kernel execution pointers depends on replication_input value
  */
  matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock, 0, stream[0]>>>(d_A1, d_B1, d_C1, N);
  if (NUMBER_OF_STREAMS==2){matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock, 0, stream[1]>>>(d_pA, d_pB, d_C2, N);}

  /*
  * Sync streams 
  */
  HANDLE_ERROR (hipStreamSynchronize(stream[0]) );
  if (NUMBER_OF_STREAMS==2){HANDLE_ERROR (hipStreamSynchronize(stream[1]) ); }

  // gettimeofday(&time_end, NULL);

  /*
  * Copy back results data from device to host memory
  */
  mem_copy_to_host(h_C1,h_C2,d_C1,d_C2, SIZE*sizeof(float),&stream[0]);
  

  /*
  * Sync streams 
  */
  for (int i = 0; i < NUMBER_OF_STREAMS; ++i)
    HANDLE_ERROR(hipStreamSynchronize(stream[i])); // SC he puesto HANDLE_ERROR

  /*
  * Destroy streams 
  */
  for (int i = 0; i < NUMBER_OF_STREAMS; ++i)
    HANDLE_ERROR( hipStreamDestroy(stream[i]));

  //SC meto esta l�nea para provocar error y ver que devuelve
  //HANDLE_ERROR( cudaStreamDestroy(stream[0]));
  /*
  * Free device GPU memory
  */
  mem_free_device(d_A1,d_B1,d_A2,d_B2,d_C1,d_C2,replicate_input);


#if INJECT_FAULTS
  if (runs_counter==2 || runs_counter==40){
    inject_faults(h_C1,N);
  }
#endif
  /*
  *Start timing
  */
  gettimeofday(&time_compare1, NULL);
  // printf("MaT_C\n");
  // printMatrix( h_C1,  N);

  /*
  * Checking if the two output arrays are the same
  */
#if USE_GOLDEN
  //bool correct_C1=true;
  bool correct_C2=true;
  correct_C1= Golden_check(h_C1, N, N, N);
  correct_C2= Golden_check(h_C2, N, N, N);
#endif
  if (NUMBER_OF_STREAMS == 1) {   
    correct_C1= Golden_check(h_C1, N, N, N);}
  else {
    DMR_correct = test(h_C1,h_C2,N);           // the comparison time is long and always the same
    //bool DMR_correct = FindFirstError(h_C1,h_C2,N);   // the comparison time is lower in average
  }
  gettimeofday(&time_compare2, NULL);
  
  /*
  * Free host memory
  */
  // Esto lo hago s�lo al final del bloque
  //mem_free_host(h_A,h_B,h_C1,h_C2);
  

    gettimeofday(&time_end, NULL);
    
  #if PERFORMANCE
    //unsigned int exTime=0;
    //exTime = (unsigned int) ((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec));
    printf("Comparison time: %u us\n", (unsigned int) ((time_compare2.tv_sec * 1000000 + time_compare2.tv_usec) - (time_compare1.tv_sec * 1000000 + time_compare1.tv_usec)));
    //printf("Execution time of both kernels: %u us\n", exTime);
  #endif

#if USE_GOLDEN
  if (DMR_correct==true) { 
      if (correct_C1==false || correct_C2==false) {
          printf("ERRORS undetected\n");
          initMat4Golden(h_A, N, N);
          initMat4Golden(h_B, N, N);
          runs_werror++;
      }
  }
  else { printf("ERRORS detected by DMR\n");
  }
#else
  if (NUMBER_OF_STREAMS == 1) {
    if (correct_C1==true) {
      //printf("OK\n");
    }
    else {
      printf("ERRORS\n");
      initMat4Golden(h_A, N, N);
      initMat4Golden(h_B, N, N);
      runs_werror++;
    } 
  }
  else { //DMR
	//PRINT a SUMMARY every rblock runs
	if (runs_counter % rblock==0 && runs_counter !=0) {
		gettimeofday(&time_end, NULL);
		//TotalexTime = (unsigned int) ((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec));
		TotalexTime = (unsigned int) ((time_end.tv_sec) - (time_start.tv_sec));
		printf("TEST_CHECK:%u;RUNS_WERROR:%d; EXEC_TIME:%us\n", rblock, runs_werror, TotalexTime);
	}
	// PRINT for each run
    if (DMR_correct==true) {printf("C\n");
    }
	else {
      printf("Init Mat\n");
      initMatrices(h_A,h_B,N);
      runs_werror++;
    }
  }
#endif
runs_counter++;

}
mem_free_host(h_A,h_B,h_C1,h_C2);
// gettimeofday(&time_end, NULL);
// TotalexTime = (unsigned int) ((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec));
// TotalexTime = (unsigned int) ((time_end.tv_sec) - (time_start.tv_sec));
//printf("TEST_CHECK:%u;RUNS_WERROR:%d; EXEC_TIME:%us\n", rblock, runs_werror, TotalexTime);


  return 0;
}
